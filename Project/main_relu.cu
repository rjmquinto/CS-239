#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>

#include "util.h"
#include "relu.h"
#include <stdio.h>
#include <time.h>

void checkError() {
    hipError_t err = hipGetLastError();
    if(err) {
        printf("Error: %s\n",hipGetErrorString(err));
        exit(1);
    }
}

int numIterations = 1000;

int main()
{
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, 0);

    float *in, *out;
	long long N = readData(&in, "relu/in.dat");
	out = (float*)malloc(N * sizeof(float));

    // printf("Executing with N=%lld\n", N);

	dim3 threadsPerBlock(deviceProp.maxThreadsPerBlock);
	dim3 blocksPerGrid((N+deviceProp.maxThreadsPerBlock-1)/deviceProp.maxThreadsPerBlock);
	// printf("relu:\n");

	float *in_dev, *out_dev;
	do {
		hipMalloc(&in_dev, N*sizeof(float));
    } while(hipGetLastError());

	do {
		hipMalloc(&out_dev, N*sizeof(float));
	} while(hipGetLastError());

	do {
		hipMemcpy(in_dev, in, N*sizeof(float), hipMemcpyHostToDevice);
	} while(hipGetLastError());

	clock_t start, end;
	float totalTime = 0;

	for(int i=0; i<numIterations; i++)
    {
		start = clock();
		do {
			relu<<<blocksPerGrid, threadsPerBlock>>>(in_dev, out_dev, N);
		} while(hipGetLastError());
		do {
			hipDeviceSynchronize();
		} while(hipGetLastError());
		end = clock();

		totalTime += (end - start);
		printf("%f\n",(end-start)*1000.00/CLOCKS_PER_SEC);

		// hipMemcpy(out, out_dev, N*sizeof(float), hipMemcpyDeviceToHost);
	}

	// printf("Average time elapsed: %fms\n\n", totalTime * 1000.0 / CLOCKS_PER_SEC / numIterations);

	do {
		hipFree(in_dev);
	} while(hipGetLastError());
	do {
		hipFree(out_dev);
	} while(hipGetLastError());

	free(in);
	free(out);

    return 0;
}
