#include <hip/hip_runtime.h>
#include <>

#include "util.h"
#include "relu.h"
#include "lstm.h"
#include <stdio.h>
#include <math.h>
#include <time.h>

extern void printAllCUDASpecs(hipDeviceProp_t& deviceProp);
void initPrint();

/*
 * Matrix Multiplication
 * C = AB
 */

void runReluExperiment(hipDeviceProp_t &deviceProp)
{
	const long long N = (1ll << 25);
	printf("Executing with N=%lld\n", N);

	float *A, *B;
	A = (float*)malloc(N * sizeof(float));
	B = (float*)malloc(N * sizeof(float));

	execute_relu(A, B, N, deviceProp);
	execute_relu_serial(A, B, N);

	free(A);
	free(B);
}

void runLSTMExperiment(hipDeviceProp_t &deviceProp)
{
	const long long N = (1ll << 10);
	const long long M = (1ll << 15);
	printf("Executing with N=%lld, M=%lld\n", N, M);

	float *A, *B;
	A = (float*)malloc(N*M*sizeof(float));
	B = (float*)malloc(N*M*sizeof(float));

	execute_lstm(A, B, N, M, deviceProp);
	execute_lstm_serial(A, B, N, M);

	free(A);
	free(B);
}

int main()
{
	initPrint();
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, 0);

	runReluExperiment(deviceProp);
	runLSTMExperiment(deviceProp);

    return 0;
}

void initPrint() {
	int device;
	int deviceCount;
	hipGetDeviceCount(&deviceCount);
	for (device = 0; device < deviceCount; ++device) {
		hipDeviceProp_t deviceProp;
		hipGetDeviceProperties(&deviceProp, device);
		printf("\nDevice %d: \"%s\"\n", device, deviceProp.name);
		printAllCUDASpecs(deviceProp);
	}
}
