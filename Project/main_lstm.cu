#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>

#include "util.h"
#include "lstm.h"
#include <stdio.h>
#include <time.h>

void checkError() {
    hipError_t err = hipGetLastError();
    if(err) {
        printf("Error: %s\n",hipGetErrorString(err));
        exit(1);
    }
}

int numIterations = 1000;

int main()
{
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, 0);

    float *in, *out, *W, *B, *U;
    long long M = readData(&W,"lstm/w.dat");
    M >>= 2;
    readData(&B,"lstm/b.dat");
    readData(&U,"lstm/u.dat");
	long long N = readData(&in, "lstm/in.dat") / M;
	out = (float*)malloc(N * M * sizeof(float));

    // printf("Executing with N=%lld, M=%lld\n", N, M);

	dim3 threadsPerBlock(deviceProp.maxThreadsPerBlock);
	dim3 blocksPerGrid((M+deviceProp.maxThreadsPerBlock-1)/deviceProp.maxThreadsPerBlock);
	// printf("lstm:\n");

	float *in_dev, *out_dev, *W_dev, *B_dev, *U_dev;
    do {
	hipMalloc(&in_dev, N*M*sizeof(float));
    } while(hipGetLastError());

    do {
	    hipMalloc(&out_dev, N*M*sizeof(float));
    } while(hipGetLastError());
    do {
        hipMalloc(&W_dev, M*4*sizeof(float));
    } while(hipGetLastError());
    do {
        hipMalloc(&B_dev, M*4*sizeof(float));
    } while(hipGetLastError());
    do {
        hipMalloc(&U_dev, M*4*sizeof(float));
    } while(hipGetLastError());

    do {
	    hipMemcpy(in_dev, in, N*M*sizeof(float), hipMemcpyHostToDevice);
    } while(hipGetLastError());
    do {
        hipMemcpy(W_dev, W, M*4*sizeof(float), hipMemcpyHostToDevice);
    } while(hipGetLastError());
    do {
        hipMemcpy(B_dev, B, M*4*sizeof(float), hipMemcpyHostToDevice);
    } while(hipGetLastError());
    do {
        hipMemcpy(U_dev, U, M*4*sizeof(float), hipMemcpyHostToDevice);
    } while(hipGetLastError());

	clock_t start, end;
	float totalTime = 0;

    for(int i=0; i<numIterations; i++)
    {
		start = clock();
        do {
		    lstm<<<blocksPerGrid, threadsPerBlock>>>(in_dev, out_dev, W_dev, B_dev, U_dev, N, M);
        } while(hipGetLastError());
        do {
		    hipDeviceSynchronize();
        } while(hipGetLastError());
		end = clock();

        checkError();

		totalTime += (end - start);
        printf("%f\n",(end-start)*1000.00/CLOCKS_PER_SEC);

		// hipMemcpy(out, out_dev, N*M*sizeof(float), hipMemcpyDeviceToHost);
        checkError();
	}

	// printf("Average time elapsed: %fms\n\n", totalTime * 1000.0 / CLOCKS_PER_SEC / numIterations);

    do {
	    hipFree(in_dev);
    } while(hipGetLastError());
    do {
	    hipFree(out_dev);
    } while(hipGetLastError());
    do {
        hipFree(W_dev);
    } while(hipGetLastError());
    do {
        hipFree(B_dev);
    } while(hipGetLastError());
    do {
        hipFree(U_dev);
    } while(hipGetLastError());

	free(in);
    free(W);
    free(B);
    free(U);
	free(out);

    return 0;
}
